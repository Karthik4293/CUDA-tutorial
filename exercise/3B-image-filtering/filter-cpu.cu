#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <limits>
#include <csignal>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#include "../util/util.h"
#include "../util/png.h"

struct png_image {
    uint width, height;
    std::vector<unsigned char> pixel;

    png_image( const char fn[] ) {
        load( fn );
    }
    uint load( const char fn[] ) {
        return lodepng::decode( pixel, width, height, fn, LCT_GREY, 8 );
    }
    uint save( const char fn[] ) {
        return lodepng::encode( fn, &pixel[0], width, height, LCT_GREY, 8 );
    }
    unsigned char *data() {
        return &pixel[0];
    }
};

// host code entrance
int main( int argc, char **argv )
{
    // the grayscale image is stored in image.data(), with one value for each pixel
    png_image image( argc > 1 ? argv[1] : "original.png" );
    png_image new_image = image;

    // image.data() returns pointer to the pixel data

    #pragma omp parallel for
    for( int i = 0; i < image.height; i++ ) {
        for( int j = 0; j < image.width; j++ ) {
            float sum = 0.f;
            for( int k = 0; k < 3; k++ ) {
                for( int l = 0; l < 3; l++ ) {
                    int K = i - 1 + k;
                    int L = j - 1 + l;
                    // using 'symmetric' boundary here
                    if( K < 0 ) K = -K;
                    if( L < 0 ) L = -L;
                    if( K >= image.height ) K = 2 * image.height - K;
                    if( L >= image.width ) L = 2 * image.width - L;
                    // adding the contributions
                    sum += image.pixel[K * image.width + L] * 0.111111;
                }
            }
            new_image.pixel[i * new_image.width + j] = sum;
        }
    }

    new_image.save( "result.png" );
}
